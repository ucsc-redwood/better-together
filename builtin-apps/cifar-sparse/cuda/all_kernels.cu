#include "hip/hip_runtime.h"
#include <>

#include <cfloat>

#include "all_kernels.cuh"

namespace cifar_sparse::cuda {

// ----------------------------------------------------------------------------
// Convolution 2D (Sparse)
// ----------------------------------------------------------------------------

__global__ void conv2d(const float* input_data,
                       const int image_input_channels,
                       const int input_height,
                       const int input_width,
                       //    const CSRMatrix& weight_matrix,

                       const float* weight_matrix_values,
                       const int* weight_matrix_row_ptr,
                       const int* weight_matrix_col_idx,
                       const int weight_matrix_rows,
                       const int weight_matrix_cols,
                       const int weight_matrix_nnz,

                       const float* bias_data,
                       const int bias_size,
                       const int kernel_size,
                       const int stride,
                       const int padding,
                       const bool relu,
                       float* output_data) {
  auto thread_idx = threadIdx.x;
  auto i = blockIdx.x * blockDim.x + thread_idx;

  int output_height = (input_height + 2 * padding - kernel_size) / stride + 1;
  int output_width = (input_width + 2 * padding - kernel_size) / stride + 1;
  // int output_channels = weight_matrix.rows;
  // int spatial_size = kernel_size * kernel_size * image_input_channels;

  // // Zero initialize output
  // int output_size = output_channels * output_height * output_width;
  // for (int i = 0; i < output_size; ++i) {
  //   output_data[i] = 0.0f;
  // }

  //   for (int out_c = start; out_c < end; ++out_c) {
  if (i >= weight_matrix_rows) {
    return;
  }

  auto out_c = i;

  // for (int out_c = 0; out_c < output_channels; ++out_c) {
  int row_start = weight_matrix_row_ptr[out_c];
  int row_end = weight_matrix_row_ptr[out_c + 1];

  for (int oh = 0; oh < output_height; ++oh) {
    for (int ow = 0; ow < output_width; ++ow) {
      float sum = 0.0f;

      for (int nz_idx = row_start; nz_idx < row_end; ++nz_idx) {
        int flat_kernel_idx = weight_matrix_col_idx[nz_idx];
        float weight_value = weight_matrix_values[nz_idx];

        int in_c = flat_kernel_idx / (kernel_size * kernel_size);
        int rem = flat_kernel_idx % (kernel_size * kernel_size);
        int ky = rem / kernel_size;
        int kx = rem % kernel_size;

        int ih = oh * stride + ky - padding;
        int iw = ow * stride + kx - padding;

        if (ih >= 0 && ih < input_height && iw >= 0 && iw < input_width) {
          int input_idx = (in_c * input_height + ih) * input_width + iw;
          sum += input_data[input_idx] * weight_value;
        }
      }

      if (bias_data && out_c < bias_size) {
        sum += bias_data[out_c];
      }

      if (relu && sum < 0) {
        sum = 0.0f;
      }

      output_data[(out_c * output_height + oh) * output_width + ow] = sum;
    }
  }

  //   }
}

// // start, end = 0, weight_matrix.rows;
// __global__ void conv2d(const float* input_data,
//                        const int image_input_channels,
//                        const int input_height,
//                        const int input_width,
//                        const CSRMatrix& weight_matrix,
//                        const float* bias_data,
//                        const int bias_size,
//                        const int kernel_size,
//                        const int stride,
//                        const int padding,
//                        const bool relu,
//                        float* output_data
//                        //    const int output_height,
//                        //    const int output_width,
// ) {
//   int out_c = blockIdx.x;                          // Output channel index
//   int oh = blockIdx.y * blockDim.y + threadIdx.y;  // Output height index
//   int ow = blockIdx.z * blockDim.z + threadIdx.z;  // Output width index

//   auto output_height = (input_height + 2 * padding - kernel_size) / stride +
//   1; auto output_width = (input_width + 2 * padding - kernel_size) / stride +
//   1;

//   if (out_c >= weight_matrix.rows || oh >= output_height ||
//       ow >= output_width) {
//     return;  // Out-of-bounds check
//   }

//   int row_start = weight_matrix.row_ptr[out_c];
//   int row_end = weight_matrix.row_ptr[out_c + 1];
//   float sum = 0.0f;

//   for (int nz_idx = row_start; nz_idx < row_end; ++nz_idx) {
//     int flat_kernel_idx = weight_matrix.col_idx[nz_idx];
//     float weight_value = weight_matrix.values[nz_idx];

//     int in_c = flat_kernel_idx / (kernel_size * kernel_size);
//     int rem = flat_kernel_idx % (kernel_size * kernel_size);
//     int ky = rem / kernel_size;
//     int kx = rem % kernel_size;

//     int ih = oh * stride + ky - padding;
//     int iw = ow * stride + kx - padding;

//     if (ih >= 0 && ih < input_height && iw >= 0 && iw < input_width) {
//       int input_idx = (in_c * input_height + ih) * input_width + iw;
//       sum += input_data[input_idx] * weight_value;
//     }
//   }

//   if (bias_data && out_c < bias_size) {
//     sum += bias_data[out_c];
//   }

//   if (relu && sum < 0) {
//     sum = 0.0f;
//   }

//   int output_idx = (out_c * output_height + oh) * output_width + ow;
//   output_data[output_idx] = sum;
// }

// ----------------------------------------------------------------------------
// Max Pooling 2D (Sparse)
// ----------------------------------------------------------------------------

__global__ void maxpool2d(const float* input_data,
                          int input_channels,
                          int input_height,
                          int input_width,
                          int pool_size,
                          int stride,
                          float* output_data) {
  auto thread_idx = threadIdx.x;
  auto i = blockIdx.x * blockDim.x + thread_idx;

  int output_height = (input_height - pool_size) / stride + 1;
  int output_width = (input_width - pool_size) / stride + 1;
  // int total_iterations = input_channels * output_height * output_width;

  if (i >= input_channels * output_height * output_width) {
    return;
  }

  auto index = i;

  //   for (int index = start; index < end; index++) {
  int c = index / (output_height * output_width);
  int h = (index / output_width) % output_height;
  int w = index % output_width;

  float max_val = -FLT_MAX;
  for (int p = 0; p < pool_size * pool_size; p++) {
    int ph = p / pool_size;
    int pw = p % pool_size;

    int input_h = h * stride + ph;
    int input_w = w * stride + pw;
    if (input_h < input_height && input_w < input_width) {
      int input_index = c * (input_height * input_width) + input_h * input_width + input_w;
      max_val = max(max_val, input_data[input_index]);
    }
  }
  int output_index = c * (output_height * output_width) + h * output_width + w;
  output_data[output_index] = max_val;
  //   }
}

// ----------------------------------------------------------------------------
// Linear Layer (Sparse)
// ----------------------------------------------------------------------------

__global__ void linear(const float* input_data,
                       //    const CSRMatrix& weight_matrix,

                       const float* weight_matrix_values,
                       const int* weight_matrix_row_ptr,
                       const int* weight_matrix_col_idx,
                       const int weight_matrix_rows,
                       const int weight_matrix_cols,
                       const int weight_matrix_nnz,

                       const float* bias_data,
                       float* output_data) {
  auto thread_idx = threadIdx.x;
  auto i = blockIdx.x * blockDim.x + thread_idx;

  if (i >= weight_matrix_rows) {
    return;
  }

  //   for (int i = start; i < end; ++i) {
  //   for (int i = start; i < end; ++i) {
  float sum = 0.0f;

  for (int nz_idx = weight_matrix_row_ptr[i]; nz_idx < weight_matrix_row_ptr[i + 1]; ++nz_idx) {
    int col = weight_matrix_col_idx[nz_idx];
    sum += input_data[col] * weight_matrix_values[nz_idx];
  }

  output_data[i] = sum + bias_data[i];
  //   }
}

// ----------------------------------------------------------------------------
// v2
// ----------------------------------------------------------------------------

namespace v2 {

__global__ void conv2d_cuda_kernel(const float* input_data,
                                   int batch_size,
                                   int in_channels,
                                   int in_height,
                                   int in_width,
                                   const float* weight_vals,
                                   const int* weight_row_ptr,
                                   const int* weight_col_idx,
                                   int out_channels,
                                   const float* bias_data,
                                   int bias_size,
                                   int kernel_size,
                                   int stride,
                                   int padding,
                                   bool relu,
                                   float* output_data,
                                   int out_height,
                                   int out_width) {
  // Compute a linear index across the entire output tensor.
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int total_output = batch_size * out_channels * out_height * out_width;
  if (index >= total_output) return;

  // Decode the 1D index into 4D indices: (b, out_c, oh, ow).
  int ow = index % out_width;
  int tmp = index / out_width;
  int oh = tmp % out_height;
  tmp = tmp / out_height;
  int out_c = tmp % out_channels;
  int b = tmp / out_channels;

  float sum = 0.0f;
  // Get the start and end indices in the CSR for this output channel.
  int row_start = weight_row_ptr[out_c];
  int row_end = weight_row_ptr[out_c + 1];
  int kernel_area = kernel_size * kernel_size;

  // Loop over each nonzero weight contributing to this output channel.
  for (int nz = row_start; nz < row_end; nz++) {
    int flat_kernel_idx = weight_col_idx[nz];
    float weight_val = weight_vals[nz];

    // Decode the flat index into an input channel and kernel (ky, kx) position.
    int in_c = flat_kernel_idx / kernel_area;
    int rem = flat_kernel_idx % kernel_area;
    int ky = rem / kernel_size;
    int kx = rem % kernel_size;

    // Compute the corresponding input spatial coordinates.
    int in_y = oh * stride + ky - padding;
    int in_x = ow * stride + kx - padding;

    // If within the input boundaries, accumulate the weighted input.
    if (in_y >= 0 && in_y < in_height && in_x >= 0 && in_x < in_width) {
      int input_idx = ((b * in_channels + in_c) * in_height + in_y) * in_width + in_x;
      sum += input_data[input_idx] * weight_val;
    }
  }

  // Add bias if provided.
  if (bias_data != nullptr && out_c < bias_size) {
    sum += bias_data[out_c];
  }
  // Apply ReLU activation if needed.
  if (relu && sum < 0.0f) {
    sum = 0.0f;
  }

  // Write the computed sum to the output tensor.
  output_data[index] = sum;
}

}  // namespace v2

}  // namespace cifar_sparse::cuda
