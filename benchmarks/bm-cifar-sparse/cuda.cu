#include <benchmark/benchmark.h>

#include "../argc_argv_sanitizer.hpp"
#include "builtin-apps/app.hpp"
#include "builtin-apps/cifar-sparse/cuda/dispatchers.cuh"
#include "builtin-apps/cifar-sparse/sparse_appdata.hpp"
#include "builtin-apps/common/cuda/cu_bench_helper.cuh"
#include "builtin-apps/common/cuda/helpers.cuh"
#include "builtin-apps/common/cuda/manager.cuh"
#include "builtin-apps/resources_path.hpp"

#define PREPARE_DATA                 \
  cuda::CudaManager mgr;             \
  auto mr = &mgr.get_mr();           \
  cifar_sparse::AppData appdata(mr); \
  CheckCuda(hipDeviceSynchronize());

// ----------------------------------------------------------------
// Global config
// ----------------------------------------------------------------

bool g_flush_l2_cache = false;

// ----------------------------------------------------------------
// Baseline
// ----------------------------------------------------------------

class CUDA_CifarSparse : public benchmark::Fixture {};

BENCHMARK_DEFINE_F(CUDA_CifarSparse, Baseline)
(benchmark::State& state) {
  PREPARE_DATA;

  for (auto _ : state) {
    CudaEventTimer timer(state, g_flush_l2_cache);
    cifar_sparse::cuda::process_stage_1(appdata);
    cifar_sparse::cuda::process_stage_2(appdata);
    cifar_sparse::cuda::process_stage_3(appdata);
    cifar_sparse::cuda::process_stage_4(appdata);
    cifar_sparse::cuda::process_stage_5(appdata);
    cifar_sparse::cuda::process_stage_6(appdata);
    cifar_sparse::cuda::process_stage_7(appdata);
    cifar_sparse::cuda::process_stage_8(appdata);
    cifar_sparse::cuda::process_stage_9(appdata);
  }
}

BENCHMARK_REGISTER_F(CUDA_CifarSparse, Baseline)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 1
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_CifarSparse, Stage1)
(benchmark::State& state) {
  PREPARE_DATA;

  // previous steps + warmup
  cifar_sparse::cuda::process_stage_1(appdata);
  CheckCuda(hipDeviceSynchronize());

  for (auto _ : state) {
    CudaEventTimer timer(state, g_flush_l2_cache);
    cifar_sparse::cuda::process_stage_1(appdata);
  }
}

BENCHMARK_REGISTER_F(CUDA_CifarSparse, Stage1)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 2
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_CifarSparse, Stage2)
(benchmark::State& state) {
  PREPARE_DATA;

  cifar_sparse::cuda::process_stage_1(appdata);
  cifar_sparse::cuda::process_stage_2(appdata);
  CheckCuda(hipDeviceSynchronize());

  for (auto _ : state) {
    CudaEventTimer timer(state, g_flush_l2_cache);
    cifar_sparse::cuda::process_stage_2(appdata);
  }
}

BENCHMARK_REGISTER_F(CUDA_CifarSparse, Stage2)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 3
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_CifarSparse, Stage3)
(benchmark::State& state) {
  PREPARE_DATA;

  cifar_sparse::cuda::process_stage_1(appdata);
  cifar_sparse::cuda::process_stage_2(appdata);
  cifar_sparse::cuda::process_stage_3(appdata);
  CheckCuda(hipDeviceSynchronize());

  for (auto _ : state) {
    CudaEventTimer timer(state, g_flush_l2_cache);
    cifar_sparse::cuda::process_stage_3(appdata);
  }
}

BENCHMARK_REGISTER_F(CUDA_CifarSparse, Stage3)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 4
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_CifarSparse, Stage4)
(benchmark::State& state) {
  PREPARE_DATA;

  cifar_sparse::cuda::process_stage_1(appdata);
  cifar_sparse::cuda::process_stage_2(appdata);
  cifar_sparse::cuda::process_stage_3(appdata);
  cifar_sparse::cuda::process_stage_4(appdata);
  CheckCuda(hipDeviceSynchronize());

  for (auto _ : state) {
    CudaEventTimer timer(state, g_flush_l2_cache);
    cifar_sparse::cuda::process_stage_4(appdata);
  }
}

BENCHMARK_REGISTER_F(CUDA_CifarSparse, Stage4)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 5
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_CifarSparse, Stage5)
(benchmark::State& state) {
  PREPARE_DATA;

  cifar_sparse::cuda::process_stage_1(appdata);
  cifar_sparse::cuda::process_stage_2(appdata);
  cifar_sparse::cuda::process_stage_3(appdata);
  cifar_sparse::cuda::process_stage_4(appdata);
  cifar_sparse::cuda::process_stage_5(appdata);
  CheckCuda(hipDeviceSynchronize());

  for (auto _ : state) {
    CudaEventTimer timer(state, g_flush_l2_cache);
    cifar_sparse::cuda::process_stage_5(appdata);
  }
}

BENCHMARK_REGISTER_F(CUDA_CifarSparse, Stage5)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 6
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_CifarSparse, Stage6)
(benchmark::State& state) {
  PREPARE_DATA;

  cifar_sparse::cuda::process_stage_1(appdata);
  cifar_sparse::cuda::process_stage_2(appdata);
  cifar_sparse::cuda::process_stage_3(appdata);
  cifar_sparse::cuda::process_stage_4(appdata);
  cifar_sparse::cuda::process_stage_5(appdata);
  cifar_sparse::cuda::process_stage_6(appdata);
  CheckCuda(hipDeviceSynchronize());

  for (auto _ : state) {
    CudaEventTimer timer(state, g_flush_l2_cache);
    cifar_sparse::cuda::process_stage_6(appdata);
  }
}

BENCHMARK_REGISTER_F(CUDA_CifarSparse, Stage6)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 7
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_CifarSparse, Stage7)
(benchmark::State& state) {
  PREPARE_DATA;

  cifar_sparse::cuda::process_stage_1(appdata);
  cifar_sparse::cuda::process_stage_2(appdata);
  cifar_sparse::cuda::process_stage_3(appdata);
  cifar_sparse::cuda::process_stage_4(appdata);
  cifar_sparse::cuda::process_stage_5(appdata);
  cifar_sparse::cuda::process_stage_6(appdata);
  cifar_sparse::cuda::process_stage_7(appdata);
  CheckCuda(hipDeviceSynchronize());

  for (auto _ : state) {
    CudaEventTimer timer(state, g_flush_l2_cache);
    cifar_sparse::cuda::process_stage_7(appdata);
  }
}

BENCHMARK_REGISTER_F(CUDA_CifarSparse, Stage7)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 8
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_CifarSparse, Stage8)
(benchmark::State& state) {
  PREPARE_DATA;

  cifar_sparse::cuda::process_stage_1(appdata);
  cifar_sparse::cuda::process_stage_2(appdata);
  cifar_sparse::cuda::process_stage_3(appdata);
  cifar_sparse::cuda::process_stage_4(appdata);
  cifar_sparse::cuda::process_stage_5(appdata);
  cifar_sparse::cuda::process_stage_6(appdata);
  cifar_sparse::cuda::process_stage_7(appdata);
  cifar_sparse::cuda::process_stage_8(appdata);
  CheckCuda(hipDeviceSynchronize());

  for (auto _ : state) {
    CudaEventTimer timer(state, g_flush_l2_cache);
    cifar_sparse::cuda::process_stage_8(appdata);
  }
}

BENCHMARK_REGISTER_F(CUDA_CifarSparse, Stage8)->Unit(benchmark::kMillisecond);

// ----------------------------------------------------------------
// Stage 9
// ----------------------------------------------------------------

BENCHMARK_DEFINE_F(CUDA_CifarSparse, Stage9)
(benchmark::State& state) {
  PREPARE_DATA;

  cifar_sparse::cuda::process_stage_1(appdata);
  cifar_sparse::cuda::process_stage_2(appdata);
  cifar_sparse::cuda::process_stage_3(appdata);
  cifar_sparse::cuda::process_stage_4(appdata);
  cifar_sparse::cuda::process_stage_5(appdata);
  cifar_sparse::cuda::process_stage_6(appdata);
  cifar_sparse::cuda::process_stage_7(appdata);
  cifar_sparse::cuda::process_stage_8(appdata);
  cifar_sparse::cuda::process_stage_9(appdata);
  CheckCuda(hipDeviceSynchronize());

  for (auto _ : state) {
    CudaEventTimer timer(state, g_flush_l2_cache);
    cifar_sparse::cuda::process_stage_9(appdata);
  }
}

BENCHMARK_REGISTER_F(CUDA_CifarSparse, Stage9)->Unit(benchmark::kMillisecond);

int main(int argc, char** argv) {
  PARSE_ARGS_BEGIN

  app.add_option("--flush-l2-cache", g_flush_l2_cache, "Flush L2 cache");

  PARSE_ARGS_END

  spdlog::set_level(spdlog::level::off);

  // Where to save the results json file?
  const auto storage_location = helpers::get_benchmark_storage_location();
  const auto out_name = storage_location.string() + "/BM_CifarSparse_CUDA_" + g_device_id + ".json";

  // Sanitize the arguments to pass to Google Benchmark
  auto [new_argc, new_argv] = sanitize_argc_argv_for_benchmark(argc, argv, out_name);

  benchmark::Initialize(&new_argc, new_argv.data());
  if (benchmark::ReportUnrecognizedArguments(new_argc, new_argv.data())) return 1;
  benchmark::RunSpecifiedBenchmarks();
  benchmark::Shutdown();

  return 0;
}